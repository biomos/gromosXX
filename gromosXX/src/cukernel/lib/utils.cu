/**
 * @file utils.cu
 * implementation of utils
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include "utils.h"

#undef MODULE
#undef SUBMODULE
#define MODULE cukernel
#define SUBMODULE utils

int cukernel::check_error(const char * err_msg) {
#ifndef NDEBUG
  hipDeviceSynchronize();
#endif
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
    std::cout << "CUDA-ERROR " << err_msg << ": " << hipGetErrorString(error) << std::endl;
  return (int) error;
}

