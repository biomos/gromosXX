/**
 * @file utils.cu
 * implementation of utils
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include "utils.h"

#undef MODULE
#undef SUBMODULE
#define MODULE cuda
#define SUBMODULE utils

int cudakernel::check_error(const char * err_msg) {
#ifndef NDEBUG
  hipDeviceSynchronize();
#endif
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
    std::cout << "CUDA-ERROR " << err_msg << ": " << hipGetErrorString(error) << std::endl;
  return (int) error;
}

