/**
 * @file utils.cu
 * implementation of utils
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include "utils.h"

#undef MODULE
#undef SUBMODULE
#define MODULE cuda
#define SUBMODULE utils

int cudakernel::checkError(const char * err_msg) {
#ifndef NDEBUG
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
    std::cout << "CUDA-ERROR " << err_msg << ": " << hipGetErrorString(error) << std::endl;
  return (int) error;
#else
  return 0;
#endif
}

