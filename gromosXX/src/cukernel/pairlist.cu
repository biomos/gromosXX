#include "hip/hip_runtime.h"
/**
 * @file pairlist.cu
 * pairlist compuation
 */

#include <iostream>
#include "gpu_status.h"

#include "lib/math.h"
#include "../util/debug.h"

#undef MODULE
#undef SUBMODULE
#define MODULE interaction
#define SUBMODULE cuda


#define NUM_THREADS_PER_BLOCK 96


void cudakernel::free_pairlist(pairlist &pl) {
  hipFree(pl.list); hipFree(pl.num_neighbors); hipFree(pl.overflow);
  DEBUG(4,"Pairlist: freed pairlist")
}
void cudakernel::allocate_pairlist(pairlist &pl, unsigned int size, unsigned int max_neighbors) {
  size_t pitch;
  // allocate the number of neighbors
  hipMalloc((void**) & pl.num_neighbors, size * sizeof (unsigned int));
  // Set the memory to 0
  hipMemset(pl.num_neighbors, 0, size * sizeof (unsigned int));
  // allocate the neighbor list
  hipMallocPitch((void**)((void*)&pl.list), &pitch, size * sizeof(unsigned int), max_neighbors);
  pl.max_size = max_neighbors;
  pl.pitch = (int)pitch / sizeof(int);
  // allocate the overflow flag and set it to false
  hipMalloc((void**) &pl.overflow, sizeof(bool));
  // Set the memory to 0
  hipMemset(pl.overflow, 0, sizeof(bool));
  DEBUG(10,"Pairlist: allocated memory")
}

extern "C" void cudaCalcPairlist(gpu_status * gpu_stat) {

  unsigned int numBlocks = (unsigned int) gpu_stat->host_parameter.num_solvent_mol / ( NUM_THREADS_PER_BLOCK * gpu_stat->host_parameter.num_of_gpus ) + 1;
  dim3 dimGrid(numBlocks, 1);
  dim3 dimBlock(NUM_THREADS_PER_BLOCK, 1);

  DEBUG(10,"Pairlist: GPU ID: " << gpu_stat->host_parameter.gpu_id << " of " << gpu_stat->host_parameter.num_of_gpus
            <<  ". Blocks: " << numBlocks)
  bool overflow;
  do {
    overflow = false;
    // calculate the pairlist
    kernel_CalcPairlist <<<dimGrid, dimBlock >>>(gpu_stat->dev_parameter,
            gpu_stat->dev_pos, 
            gpu_stat->dev_pl_short, gpu_stat->dev_pl_long,
            gpu_stat->host_parameter.num_of_gpus, gpu_stat->host_parameter.gpu_id);

    
    hipDeviceSynchronize();
    DEBUG(10,"Pairlist: Executed kernel and synchronized Threads")

    bool overflow_short, overflow_long;
    // get the overflow flags
    hipMemcpy(&overflow_short, gpu_stat->dev_pl_short.overflow, sizeof (bool), hipMemcpyDeviceToHost);
    hipMemcpy(&overflow_long, gpu_stat->dev_pl_long.overflow, sizeof (bool), hipMemcpyDeviceToHost);

    //unsigned int num_neighbors[gpu_stat->host_parameter.num_solvent_mol];
    //hipMemcpy(num_neighbors, gpu_stat->dev_pl_short.num_neighbors, gpu_stat->host_parameter.num_solvent_mol*sizeof (unsigned int), hipMemcpyDeviceToHost);

    // DEBUGGING
    //for(unsigned int i = 0; i < gpu_stat->host_parameter.num_solvent_mol; ++i) {
    //  if (gpu_stat->host_parameter.gpu_id==0)
    //  DEBUG(15,"nn " << i << ":" << num_neighbors[i]);
    //}

    // guard for overflow
    if (overflow_short) {
      overflow = true;
      // add 20% more space.
      DEBUG(1,"short overflow");
      DEBUG(1,"max_size = " << gpu_stat->dev_pl_short.max_size);
      const unsigned int new_estimate = gpu_stat->dev_pl_short.max_size + int(gpu_stat->dev_pl_short.max_size * 0.2f);
      free_pairlist(gpu_stat->dev_pl_short);
      allocate_pairlist(gpu_stat->dev_pl_short, gpu_stat->host_parameter.num_solvent_mol, new_estimate);
    }
    if (overflow_long) {
      overflow = true;
      // add 20% more space.
      DEBUG(1,"long overflow");
      DEBUG(1,"max_size = " << gpu_stat->dev_pl_long.max_size);
      const unsigned int new_estimate = gpu_stat->dev_pl_long.max_size + int(gpu_stat->dev_pl_long.max_size * 0.2f);
      free_pairlist(gpu_stat->dev_pl_long);
      allocate_pairlist(gpu_stat->dev_pl_long, gpu_stat->host_parameter.num_solvent_mol, new_estimate);
    }

    // warn the user that this is an issue
    if (overflow) {
      std::cout << "CUDA: Overflow. Recalculating pairlist. This is a performance issue "
              "increase size estimate." << std::endl;
    }
  } while (overflow); // recalculate the pairlist
  
}


__global__ void cudakernel::kernel_CalcPairlist
(
        cudakernel::simulation_parameter * dev_params,
        float3 * dev_pos,
        pairlist pl_short,
        pairlist pl_long,
        unsigned int num_of_gpus,
        unsigned int gpu_id
) {

  unsigned int num_neighbors_long = 0, num_neighbors_short = 0;
  __shared__ float shared_pos[NUM_THREADS_PER_BLOCK * 3];

  // take host_parameter local
  const unsigned int N = dev_params->num_atoms;
  const unsigned int num_solvent_mol = dev_params->num_solvent_mol;
  const float cutoff_long_2 = dev_params->cutoff_long_2;
  const float cutoff_short_2 = dev_params->cutoff_short_2;
  //box edges
  const float box_x = dev_params->box_x;
  const float box_y = dev_params->box_y;
  const float box_z = dev_params->box_z;
  
  const float box_inv_x = dev_params->box_inv_x;
  const float box_inv_y = dev_params->box_inv_y;
  const float box_inv_z = dev_params->box_inv_z;

  const unsigned int solvent_offset = dev_params->num_atoms_per_mol;

  // calculate indices
  const unsigned int index = blockIdx.x * NUM_THREADS_PER_BLOCK + threadIdx.x;
  const unsigned int molecule_index = index * num_of_gpus + gpu_id;

  const unsigned int first_atom_index = molecule_index*solvent_offset;
  const unsigned int myThreadOffset = threadIdx.x*solvent_offset;

  float3 first_atom_pos;
  if (first_atom_index < N)
    first_atom_pos = dev_pos[first_atom_index];

  for (unsigned int i = 0; i < N; i += (NUM_THREADS_PER_BLOCK * solvent_offset)) {
    float3 neighbor_pos;
    if (i + myThreadOffset < N)
      neighbor_pos = dev_pos[i + myThreadOffset];

    // cache a block of positions
    __syncthreads();
    shared_pos[threadIdx.x] = neighbor_pos.x;
    shared_pos[threadIdx.x + NUM_THREADS_PER_BLOCK] = neighbor_pos.y;
    shared_pos[threadIdx.x + 2 * NUM_THREADS_PER_BLOCK] = neighbor_pos.z;
    __syncthreads();

    unsigned int end_i_loop = NUM_THREADS_PER_BLOCK;
    if (end_i_loop > (N - i) / solvent_offset)
      end_i_loop = (N - i) / solvent_offset;

    if (first_atom_index < N) {
      for (unsigned int start_i_loop = 0; start_i_loop < end_i_loop; start_i_loop++) {
        const unsigned int current_first_atom_index = i + start_i_loop*solvent_offset;
        if (current_first_atom_index != first_atom_index && current_first_atom_index < N) {
          //{ calculate distance and d^2
          float3 dist;
          dist.x = first_atom_pos.x - shared_pos[start_i_loop];
          dist.x -= box_x * rintf(dist.x * box_inv_x);
          dist.y = first_atom_pos.y - shared_pos[start_i_loop + NUM_THREADS_PER_BLOCK];
          dist.y -= box_y * rintf(dist.y * box_inv_y);
          dist.z = first_atom_pos.z - shared_pos[start_i_loop + 2 * NUM_THREADS_PER_BLOCK];
          dist.z -= box_z * rintf(dist.z * box_inv_z);
          const float d2 = abs2(dist);
          //} calculate distance and d^2
       // are they interacting?
       if (d2 < cutoff_long_2) {
            // longrange?
            if (d2 > cutoff_short_2) {
              if (num_neighbors_long < pl_long.max_size) {
                pl_long.list[index + pl_long.pitch * num_neighbors_long] = current_first_atom_index;
                num_neighbors_long++;
              } else {
                *pl_long.overflow = true;
              } // overflow
            } else { // shortrange then
              if (num_neighbors_short < pl_short.max_size) {
                pl_short.list[index + pl_short.pitch * num_neighbors_short] = current_first_atom_index;
                num_neighbors_short++;
              } else {
                *pl_short.overflow = true;
              } // overflow
            } // if shortrange / longrange
          } // if cutoff
        } // if atom in valid range
      } // for atoms j
    } // if atom in valid range
  } // for atoms i
  if (molecule_index < num_solvent_mol) {
    pl_long.num_neighbors[index] = num_neighbors_long;
    pl_short.num_neighbors[index] = num_neighbors_short;
  }
}

#undef DEBUG

