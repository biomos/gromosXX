#include "hip/hip_runtime.h"
/*
 * This file is part of GROMOS.
 * 
 * Copyright (c) 2011, 2012, 2016, 2018, 2021, 2023 Biomos b.v.
 * See <https://www.gromos.net> for details.
 * 
 * GROMOS is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 2 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <https://www.gnu.org/licenses/>.
 */

/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id: ParticleData.cu 1826 2009-04-27 22:18:31Z joaander $
// $URL: https://codeblue.umich.edu/hoomd-blue/svn/tags/hoomd-0.8.2/src/cuda/ParticleData.cu $
// Maintainer: joaander

#include "ParticleData.cuh"
#include "gpu_settings.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file ParticleData.cu
 	\brief Contains GPU kernel code and data structure functions used by ParticleData
*/

//! Kernel for un-interleaving float4 input into float output
/*! \param d_out Device pointer to write un-interleaved output
	\param d_in Device pointer to read interleaved input
	\param N Number of elements in input
	\param pitch Spacing of arrays through the output

	\pre N/block_size + 1 blocks are run on the device
*/
extern "C" __global__ void uninterleave_float4_kernel(float *d_out, float4 *d_in, int N, int pitch)
    {
	int pidx  = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (pidx < N)
		{
		float4 in = d_in[pidx];
		
		d_out[pidx] = in.x;
		d_out[pidx+pitch] = in.y;
		d_out[pidx+pitch+pitch] = in.z;
		d_out[pidx+pitch+pitch+pitch] = in.w;
		}
	}


/*! The most efficient data storage on the device is to put x,y,z,type into a float4
	data structure. The most efficient storage on the CPU is x,y,z,type each as 
	separate arrays. Translation between the two is best done on the device, and
	memory transfers done with one big hipMemcpy. This function, and its sister
	gpu_interleave_float4() perform the transformation between a float* with x,y,z,type
	packed non-interleaved to a float4* storing the same values interleaved. 

	Performance is best when pitch is a multiple of 64.

	\param d_out Device pointer to write output to
	\param d_in Device pointer to read input from
	\param N Number of elements to interleave
	\param pitch Spacing between \c x[0] and \c y[0] in \a d_out

	\post A code snipped best describes what is done:
	\verbatim 
	d_out[i] = d_in[i].x
	d_out[i+pitch] = d_in[i].y
	d_out[i+pitch*2] = d_in[i].z
	d_out[i+pitch*3] = d_in[i].w
	\endverbatim

	\returns Any error code from the kernel call retrieved via hipGetLastError()
	\note Always returns hipSuccess in release builds for performance reasons
*/
hipError_t gpu_uninterleave_float4(float *d_out, float4 *d_in, int N, int pitch)
	{
	assert(pitch >= N);
	assert(d_out);
	assert(d_in);
	assert(N > 0);

	const int M = 64;
	uninterleave_float4_kernel<<< N/M+1, M >>>(d_out, d_in, N, pitch);

	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

//! Kernel for interleaving float input into float4 output
/*! \param d_out Device pointer to write interleaved output
	\param d_in Device pointer to read non-interleaved input
	\param N Number of elements in output
	\param pitch Spacing of arrays through the input

	\pre N/block_size + 1 blocks are run on the device
*/
extern "C" __global__ void interleave_float4_kernel(float4 *d_out, float *d_in, int N, int pitch)
    {
    int pidx  = blockDim.x * blockIdx.x + threadIdx.x;

    if (pidx < N)
        {
        float x = d_in[pidx];
        float y = d_in[pidx+pitch];
        float z = d_in[pidx+pitch+pitch];
        float w = d_in[pidx+pitch+pitch+pitch];

        float4 out;
        out.x = x;
        out.y = y;
        out.z = z;
        out.w = w;
        d_out[pidx] = out;
        }
    }

/*! See gpu_uninterleave_float4() for details.
	\param d_out Device pointer to write output to
	\param d_in Device pointer to read input from
	\param N Number of elements to interleave
	\param pitch Spacing between \c x[0] and \c y[0] in \a d_in
	
	\returns Any error code from the kernel call retrieved via hipGetLastError()
	\note Always returns hipSuccess in release builds for performance reasons
*/
hipError_t gpu_interleave_float4(float4 *d_out, float *d_in, int N, int pitch)
	{
	assert(pitch >= N);
	assert(d_out);
	assert(d_in);
	assert(N > 0);

	const int M = 64;
	interleave_float4_kernel<<< N/M+1, M >>>(d_out, d_in, N, pitch);

	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}
 	
////////////////////////////////////////////////////////////////////
// Unit testing functions

//! Kernel for filling \a pdata with nonsense numbers for test purposes
/*! \param pdata Particle data to populate
*/
__global__ void pdata_test_fill(gpu_pdata_arrays pdata)
	{
	// start by identifying the particle index of this particle
	int pidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pidx < pdata.N)
		{
		// write out a nonsense test pattern
		float4 pos;
		pos.x = float(pidx);
		pos.y = float(pidx)*0.5f;
		pos.z = float(pidx)*0.4f;
		pos.w = float(pidx)*0.2f;
		pdata.pos[pidx] = pos;
		
		float4 vel;
		vel.x = float(pidx)*10.0f;
		vel.y = float(pidx)*5.0f;
		vel.z = float(pidx)*4.0f;
		pdata.vel[pidx] = vel;
		
		float4 accel;
		accel.x = float(pidx)*20.0f;
		accel.y = float(pidx)*15.0f;
		accel.z = float(pidx)*14.0f;
		pdata.accel[pidx] = accel;
		
		pdata.tag[pidx] = pidx*30;
		pdata.rtag[pidx] = pidx*40;
		}
	}

/*! \param pdata Particle data where the arrays will be populated with garbage
	\post Device memory is filled out with a nonsense test pattern
 	Read the pdata_test_fill() code to see what the pattern is

	\returns Error result from the kernel call
*/ 
hipError_t gpu_generate_pdata_test(gpu_pdata_arrays *pdata)
	{
	assert(pdata);
	
	// setup the grid to run the kernel
	int M = 128;
	dim3 grid(pdata->N/M+1, 1, 1);
	dim3 threads(M, 1, 1);
	
	// run the kernel
	pdata_test_fill<<< grid, threads >>>(*pdata);
	hipDeviceSynchronize();
	return hipGetLastError();
	}

//! Texture for reading particle positions
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;

//! Kernel for testing texture read capability
/*! \param pdata Particle data to write to
	\post \c pdata.vel[i] holds the value read from \c pdata_pos_tex at location \c i
*/
__global__ void pdata_texread_test(gpu_pdata_arrays pdata)
	{
	// start by identifying the particle index of this particle
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int pidx = idx + pdata.local_beg;

	if (idx < pdata.local_num)
		{
		float4 pos = tex1Dfetch(pdata_pos_tex, pidx);
		pdata.vel[pidx] = pos;
		}
	}

/*!	\param pdata Particle data arrays to write the velocity 
	\pre The texture that the caller wants read from was bound with gpu_bind_pdata_textures()
	\post The vel device memory is filled out with what is read from the position texture
	\note Designed to be used for unit testing texture reads

	\returns Error result from the kernel call
*/
hipError_t gpu_pdata_texread_test(const gpu_pdata_arrays &pdata)
	{	
	// setup the grid to run the kernel
	int M = 128;
	dim3 grid(pdata.local_num/M+1, 1, 1);
	dim3 threads(M, 1, 1);

	// bind the textures
	hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;
	
	// run the kernel
	pdata_texread_test<<< grid, threads >>>(pdata);
	hipDeviceSynchronize();
	return hipGetLastError();
	}

