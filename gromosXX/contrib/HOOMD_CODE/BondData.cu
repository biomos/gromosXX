#include "hip/hip_runtime.h"
/*
 * This file is part of GROMOS.
 * 
 * Copyright (c) 2011, 2012, 2016, 2018, 2021, 2023 Biomos b.v.
 * See <https://www.gromos.net> for details.
 * 
 * GROMOS is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 2 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <https://www.gnu.org/licenses/>.
 */

/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id: BondData.cu 1826 2009-04-27 22:18:31Z joaander $
// $URL: https://codeblue.umich.edu/hoomd-blue/svn/tags/hoomd-0.8.2/src/cuda/BondData.cu $
// Maintainer: joaander

#include "BondData.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file BondData.cu
 	\brief Defines the data structures for storing bonds on the GPU.
*/

/*! \pre no allocations have been performed or deallocate() has been called after a previous allocate()
	\post Memory for \a n_bonds and \a bonds is allocated on the device
	\param num_local Number of particles local to the GPU on which this is being called
	\param alloc_height Number of bonds to allocate for each particle
	\note allocate() \b must be called on the GPU it is to allocate data on
*/
hipError_t gpu_bondtable_array::allocate(unsigned int num_local, unsigned int alloc_height)
	{
	// sanity checks
	assert(n_bonds == NULL);
	assert(bonds == NULL);
	
	height = alloc_height;
		
	// allocate n_bonds and check for errors
	hipError_t error = hipMalloc((void**)((void*)&n_bonds), num_local*sizeof(unsigned int));
	if (error != hipSuccess)
		return error;
	
	error = hipMemset((void*)n_bonds, 0, num_local*sizeof(unsigned int));
	if (error != hipSuccess)
		return error;
	
	// hipMallocPitch fails to work for coalesced reads here (dunno why), need to calculate pitch ourselves
	// round up to the nearest multiple of 32
	pitch = (num_local + (32 - num_local & 31));
	error = hipMalloc((void**)((void*)&bonds), pitch * height * sizeof(uint2));
	if (error != hipSuccess)
		return error;	
	
	error = hipMemset((void*)bonds, 0, pitch * height * sizeof(uint2));
	if (error != hipSuccess)
		return error;
		
	// all done, return success
	return hipSuccess;	
	}
	
/*! \pre allocate() has been called
	\post Memory for \a n_bonds and \a bonds is freed on the device
	\note deallocate() \b must be called on the same GPU as allocate()
*/
hipError_t gpu_bondtable_array::deallocate()
	{
	// sanity checks
	assert(n_bonds != NULL);
	assert(bonds != NULL);
	
	// free the memory
	hipError_t error = hipFree((void*)n_bonds);
	n_bonds = NULL;
	if (error != hipSuccess)
		return error;
		
	error = hipFree((void*)bonds);
	bonds = NULL;
	if (error != hipSuccess)
		return error;
	
	// all done, return success
	return hipSuccess;
	}
