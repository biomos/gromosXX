#include "hip/hip_runtime.h"
/*
 * This file is part of GROMOS.
 * 
 * Copyright (c) 2011, 2012, 2016, 2018, 2021, 2023 Biomos b.v.
 * See <https://www.gromos.net> for details.
 * 
 * GROMOS is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 2 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <https://www.gnu.org/licenses/>.
 */

/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id: NeighborListNsqGPU.cu 1983 2009-07-24 18:04:33Z akohlmey $
// $URL: https://codeblue.umich.edu/hoomd-blue/svn/tags/hoomd-0.8.2/src/cuda/NeighborListNsqGPU.cu $
// Maintainer: joaander

#include "NeighborListNsqGPU.cuh"
#include "ParticleData.cuh"
#include "gpu_settings.h"

#include <stdio.h>

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file NeighborListNsqGPU.cu
	\brief Defines data structures and methods used by NeighborListNsqGPU
*/

//! Compile time determined block size for the NSQ neighbor list calculation
const int NLIST_BLOCK_SIZE = 128;

//! Generate the neighbor list on the GPU in O(N^2) time
/*! \param nlist Neighbor list to write out
	\param pdata Particles to generate the neighbor list from
	\param box Box dimensions for handling periodic boundary conditions
	\param r_maxsq Precalculated value for r_max*r_max
	
	each thread is to compute the neighborlist for a single particle i
	each block will load a bunch of particles into shared mem and then each thread will compare it's particle
	to each particle in shmem to see if they are a neighbor. Since all threads in the block access the same 
	shmem element at the same time, the value is broadcast and there are no bank conflicts
	
	the way this funciton loads data, all data arrays need to be padded so they have a multiple of 
	blockDim.x elements. 	
*/
extern "C" __global__ void gpu_compute_nlist_nsq_kernel(gpu_nlist_array nlist, gpu_pdata_arrays pdata, gpu_boxsize box, float r_maxsq)
	{
	// shared data to store all of the particles we compare against
	__shared__ float sdata[NLIST_BLOCK_SIZE*4];
	
	// load in the particle
	int pidx = blockIdx.x * NLIST_BLOCK_SIZE + threadIdx.x;

	float4 pos = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	if (pidx < pdata.N)
		pos = pdata.pos[pidx];
		
	float px = pos.x;
	float py = pos.y;
	float pz = pos.z;

	// track the number of neighbors added so far
	int n_neigh = 0;
	
	uint4 exclude = make_uint4(0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff);
	if (pidx < pdata.N)
		exclude = nlist.exclusions[pidx];
#if defined(LARGE_EXCLUSION_LIST)
	uint4 exclude2 = make_uint4(0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff);
	uint4 exclude3 = make_uint4(0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff);
	uint4 exclude4 = make_uint4(0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff);
	if (pidx < pdata.N)
		{
		exclude2 = nlist.exclusions2[pidx];
		exclude3 = nlist.exclusions3[pidx];
		exclude4 = nlist.exclusions4[pidx];
		}
#endif
	
	// each block is going to loop over all N particles (this assumes memory is padded to a multiple of blockDim.x)
	// in blocks of blockDim.x
	for (int start = 0; start < pdata.N; start += NLIST_BLOCK_SIZE)
		{
		// load data
		float4 neigh_pos = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		if (start + threadIdx.x < pdata.N)
			neigh_pos = pdata.pos[start + threadIdx.x];
		
		// make sure everybody is caught up before we stomp on the memory
		__syncthreads();
		sdata[threadIdx.x] = neigh_pos.x;
		sdata[threadIdx.x + NLIST_BLOCK_SIZE] = neigh_pos.y;
		sdata[threadIdx.x + 2*NLIST_BLOCK_SIZE] = neigh_pos.z;
		sdata[threadIdx.x + 3*NLIST_BLOCK_SIZE] = neigh_pos.w; //< unused, but try to get compiler to fully coalesce reads

		// ensure all data is loaded
		__syncthreads();

		// now each thread loops over every particle in shmem, but doesn't loop past the end of the particle list (since
		// the block might extend that far)
		int end_offset= NLIST_BLOCK_SIZE;
		end_offset = min(end_offset, pdata.N - start);

		if (pidx < pdata.N)
			{ 

			for (int cur_offset = 0; cur_offset < end_offset; cur_offset++)
				{
				// calculate dr
				float dx = px - sdata[cur_offset];
				dx = dx - box.Lx * rintf(dx * box.Lxinv);
				
				if (dx*dx < r_maxsq)
					{
					float dy = py - sdata[cur_offset + NLIST_BLOCK_SIZE];
					dy = dy - box.Ly * rintf(dy * box.Lyinv);
				
					if (dy*dy < r_maxsq)
						{
						float dz = pz - sdata[cur_offset + 2*NLIST_BLOCK_SIZE];
						dz = dz - box.Lz * rintf(dz * box.Lzinv);
				
						float drsq = dx*dx + dy*dy + dz*dz;

#if !defined(LARGE_EXCLUSION_LIST)
						// we don't add if we are comparing to ourselves, and we don't add if we are above the cut
						if ((drsq < r_maxsq) && ((start + cur_offset) != pidx) && exclude.x != (start + cur_offset) && exclude.y != (start + cur_offset) && exclude.z != (start + cur_offset) && exclude.w != (start + cur_offset))
							{
							if (n_neigh < nlist.height)
								{
								nlist.list[pidx + n_neigh*nlist.pitch] = start+cur_offset;
								n_neigh++;
								}
							else
								*nlist.overflow = 1;
							}
#else					
						if ((drsq < r_maxsq) && ((start + cur_offset) != pidx) && exclude.x != (start + cur_offset) && exclude.y != (start + cur_offset) 
							&& exclude.z != (start + cur_offset) && exclude.w != (start + cur_offset) && exclude2.x != (start + cur_offset) 
							&& exclude2.y != (start + cur_offset) && exclude2.z != (start + cur_offset) && exclude2.w != (start + cur_offset)
							&& exclude3.x != (start + cur_offset) && exclude3.y != (start + cur_offset) && exclude3.z != (start + cur_offset) 
							&& exclude3.w != (start + cur_offset) && exclude4.x != (start + cur_offset) && exclude4.y != (start + cur_offset) 
							&& exclude4.z != (start + cur_offset) && exclude4.w != (start + cur_offset))
							{
							if (n_neigh < nlist.height)
								{
								nlist.list[pidx + n_neigh*nlist.pitch] = start+cur_offset;
								n_neigh++;
								}
							else
								*nlist.overflow = 1;
							}
#endif
						}
					}
				}
			}
		}

	// now that we are done: update the first row that lists the number of neighbors
	if (pidx < pdata.N)
		{
		nlist.n_neigh[pidx] = n_neigh;
		nlist.last_updated_pos[pidx] = pdata.pos[pidx];
		}
	}

//! Generate the neighbor list on the GPU in O(N^2) time
/*! \param nlist Neighbor list to write out
	\param pdata Particles to generate the neighbor list from
	\param box Box dimensions for handling periodic boundary conditions
	\param r_maxsq Precalculated value for r_max*r_max
		
	see generateNlistNSQ for more information
*/
hipError_t gpu_compute_nlist_nsq(const gpu_nlist_array &nlist, const gpu_pdata_arrays &pdata, const gpu_boxsize &box, float r_maxsq)
	{
	// setup the grid to run the kernel
	int M = NLIST_BLOCK_SIZE;
	dim3 grid( (pdata.N/M) + 1, 1, 1);
	dim3 threads(M, 1, 1);
	
	// zero the overflow check
	hipError_t error = hipMemset(nlist.overflow, 0, sizeof(int));
	if (error != hipSuccess)
		return error;	
	
	// run the kernel
	gpu_compute_nlist_nsq_kernel<<< grid, threads >>>(nlist, pdata, box, r_maxsq);
	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

// vim:syntax=cpp
