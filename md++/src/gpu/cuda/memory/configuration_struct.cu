#include "hip/hip_runtime.h"
/*
 * This file is part of GROMOS.
 * 
 * Copyright (c) 2011, 2012, 2016, 2018, 2021, 2023 Biomos b.v.
 * See <https://www.gromos.net> for details.
 * 
 * GROMOS is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 2 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <https://www.gnu.org/licenses/>.
 */

/**
 * @file configuration_struct.cu
 * Implementation of the light-weight configuration struct for GPU
 */

#include "stdheader.h"

#include "configuration/configuration_global.h"

#include "algorithm/algorithm.h"
#include "topology/topology.h"
#include "configuration/configuration.h"
#include "configuration/mesh.h"
#include "configuration/influence_function.h"
#include "simulation/simulation.h"
#include "simulation/multibath.h"
#include "simulation/parameter.h"

#include "math/periodicity.h"
#include "math/boundary_checks.h"
#include "util/template_split.h"

#include "configuration_struct.h"

void gpu::Configuration::copy_to_device(configuration::Configuration& conf) {
    const size_t num_atoms = conf.current().pos.size();
    using Vec = typename decltype(conf.current().pos)::value_type;

    static_assert(std::is_convertible<Vec, float3>::value,
                  "Vec must be convertible to float3");

    auto convert_and_copy = [num_atoms](const auto& src, auto& dst) {
        dst.clear();
        dst.reserve(num_atoms);
        for (const auto& v : src)
            dst.push_back(static_cast<float3>(v));
    };

    // Current
    convert_and_copy(conf.current().pos, current.pos);
    convert_and_copy(conf.current().vel, current.vel);
    convert_and_copy(conf.current().force, current.force);
    convert_and_copy(conf.current().constraint_force, current.constraint_force);

    // Old
    convert_and_copy(conf.old().pos, old.pos);
    convert_and_copy(conf.old().vel, old.vel);
    convert_and_copy(conf.old().force, old.force);
    convert_and_copy(conf.old().constraint_force, old.constraint_force);

    // copy tensors
            // Box* box;
            // float9* virial_tensor;
            // float9* kinetic_energy_tensor;
            // float9* pressure_tensor;
    Box box;
    float9 virial_tensor;
    float9 kinetic_energy_tensor;
    float9 pressure_tensor;
    
    box = conf.current().box;
    virial_tensor = conf.current().virial_tensor;
    kinetic_energy_tensor = conf.current().kinetic_energy_tensor;
    pressure_tensor = conf.current().pressure_tensor;

    hipMemcpy(current.box, &box, sizeof(box), hipMemcpyHostToDevice);
    hipMemcpy(current.virial_tensor, &virial_tensor, sizeof(virial_tensor), hipMemcpyHostToDevice);
    hipMemcpy(current.kinetic_energy_tensor, &kinetic_energy_tensor, sizeof(kinetic_energy_tensor), hipMemcpyHostToDevice);
    hipMemcpy(current.pressure_tensor, &pressure_tensor, sizeof(pressure_tensor), hipMemcpyHostToDevice);

    box = conf.old().box;
    virial_tensor = conf.old().virial_tensor;
    kinetic_energy_tensor = conf.old().kinetic_energy_tensor;
    pressure_tensor = conf.old().pressure_tensor;

    hipMemcpy(old.box, &box, sizeof(box), hipMemcpyHostToDevice);
    hipMemcpy(old.virial_tensor, &virial_tensor, sizeof(virial_tensor), hipMemcpyHostToDevice);
    hipMemcpy(old.kinetic_energy_tensor, &kinetic_energy_tensor, sizeof(kinetic_energy_tensor), hipMemcpyHostToDevice);
    hipMemcpy(old.pressure_tensor, &pressure_tensor, sizeof(pressure_tensor), hipMemcpyHostToDevice);

    current.update_view();
    old.update_view();
}