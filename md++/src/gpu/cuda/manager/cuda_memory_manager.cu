
#include <stdexcept>
#include <sstream>

#include "gpu.h"

#include "gpu/cuda/cuheader.h"
#include "gpu/cuda/utils.h"
#include "cuda_memory_manager.h"

gpu::CudaMemoryManager::CudaMemoryManager(int device_id) {}

gpu::CudaMemoryManager::~CudaMemoryManager() {}

void gpu::CudaMemoryManager::init() {
    pos_umem.resize(10000,0);
    pos_host.resize(10000,0);
}

void* gpu::CudaMemoryManager::allocate_device_memory(size_t size) {
    void* device_ptr = nullptr;
    hipSetDevice(m_device_id);
    CUDA_CHECK(hipMalloc(&device_ptr, size));
    return device_ptr;
}

void gpu::CudaMemoryManager::free_device_memory(void* device_ptr) {
    CUDA_CHECK(hipFree(device_ptr));
}

void* gpu::CudaMemoryManager::allocate_pinned_memory(size_t size) {
    void* host_ptr = nullptr;
    CUDA_CHECK(hipHostAlloc(&host_ptr, size, hipHostMallocDefault));
    return host_ptr;
}

void gpu::CudaMemoryManager::free_pinned_memory(void* host_ptr) {
    CUDA_CHECK(hipHostFree(host_ptr));
}

void gpu::CudaMemoryManager::copy_to_device(void* device_ptr, const void* host_ptr, size_t size, CUSTREAM stream) {
    CUDA_CHECK(hipMemcpy(device_ptr, host_ptr, size, hipMemcpyHostToDevice));
}

void gpu::CudaMemoryManager::copy_to_host(void* host_ptr, const void* device_ptr, size_t size, CUSTREAM stream) {
    CUDA_CHECK(hipMemcpy(host_ptr, device_ptr, size, hipMemcpyDeviceToHost));
}

void gpu::CudaMemoryManager::copy_device_to_device(void* dest_device_ptr, const void* src_device_ptr, size_t size) {
    CUDA_CHECK(hipMemcpy(dest_device_ptr, src_device_ptr, size, hipMemcpyDeviceToDevice));
}

void gpu::CudaMemoryManager::async_copy_to_device(void* device_ptr, const void* host_ptr, size_t size, hipStream_t stream) {
    CUDA_CHECK(hipMemcpyAsync(device_ptr, host_ptr, size, hipMemcpyHostToDevice, stream));
}

void gpu::CudaMemoryManager::async_copy_to_host(void* host_ptr, const void* device_ptr, size_t size, hipStream_t stream) {
    CUDA_CHECK(hipMemcpyAsync(host_ptr, device_ptr, size, hipMemcpyDeviceToHost, stream));
}

void gpu::CudaMemoryManager::query_memory(size_t& free_memory, size_t& total_memory) const {
    size_t free_mem = 0;
    size_t total_mem = 0;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    free_memory = free_mem;
    total_memory = total_mem;
}
