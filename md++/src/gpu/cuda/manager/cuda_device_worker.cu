

#include "cuda_device_worker.h"

template <typename KernelFunc, typename... Args>
void CudaDeviceWorker::launch_kernel(KernelFunc kernel, dim3 grid_dim, dim3 block_dim, Args... args, size_t shared_mem_size) {
    kernel<<<grid_dim, block_dim, shared_mem_size, stream_>>>(args...);

    // Check for errors after kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA kernel launch failed: ") + hipGetErrorString(err));
    }
}

// Explicit template instantiations (if needed)
template void CudaDeviceWorker::launch_kernel<void(*)(int*), int*>(void(*)(int*), dim3, dim3, int*, size_t);