
#include "gpu.h"

#include "cuda_device_worker.h"
#include "gpu/cuda/cuheader.h"
#include "gpu/cuda/utils.h"

gpu::CudaDeviceWorker::CudaDeviceWorker(int device_id) : m_stream(nullptr), m_device_id(device_id) {
    // Set the active device
    CUDA_CHECK(hipSetDevice(m_device_id));

    // Create a CUDA stream
    CUDA_CHECK(hipStreamCreate(&m_stream));
}

gpu::CudaDeviceWorker::~CudaDeviceWorker() {
    // Destroy the CUDA stream
    if (m_stream) {
        CUDA_CHECK(hipStreamDestroy(m_stream));
    }
}

int gpu::CudaDeviceWorker::get_device_id() const {
    return m_device_id;
}

hipStream_t gpu::CudaDeviceWorker::get_stream() const {
    return m_stream;
}

void gpu::CudaDeviceWorker::synchronize() const {
    CUDA_CHECK(hipDeviceSynchronize());
}

template <typename KernelFunc, typename... Args>
void gpu::CudaDeviceWorker::launch_kernel(KernelFunc kernel, dim3 grid_dim, dim3 block_dim, Args... args, size_t shared_mem_size) {
    kernel<<<grid_dim, block_dim, shared_mem_size, m_stream>>>(args...);

    // Check for errors after kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA kernel launch failed: ") + hipGetErrorString(err));
    }
}

// Explicit template instantiations (if needed)
// template void gpu::CudaDeviceWorker::launch_kernel<void(*)(int*), int*>(void(*)(int*), dim3, dim3, int*, size_t);

// template <typename KernelFunc, typename... Args>
// void gpu::CudaDeviceWorker::launch_kernel(KernelFunc kernel, dim3 grid_dim, dim3 block_dim, Args... args, size_t shared_mem_size) {
//     // Set the active device
//     hipSetDevice(device_id_);

//     // Launch the kernel
//     kernel<<<grid_dim, block_dim, shared_mem_size, m_stream>>>(args...);

//     // Check for kernel launch errors
//     hipError_t err = hipGetLastError();
//     if (err != hipSuccess) {
//         throw std::runtime_error("Kernel launch failed: " + std::string(hipGetErrorString(err)));
//     }
// }