

#include "cuda_device_worker.h"
#include "gpu/cuda/cuheader.h"
#include "gpu/cuda/utils.h"

gpu::CudaDeviceWorker::CudaDeviceWorker(int device_id) : device_id_(device_id), stream_(nullptr) {
    // Set the active device
    hipSetDevice(device_id_);

    // Create a CUDA stream
    hipStreamCreate(&stream_);
}

gpu::CudaDeviceWorker::~CudaDeviceWorker() {
    // Destroy the CUDA stream
    if (stream_) {
        hipStreamDestroy(stream_);
    }
}

int gpu::CudaDeviceWorker::get_device_id() const {
    return device_id_;
}

hipStream_t gpu::CudaDeviceWorker::get_stream() const {
    return stream_;
}

void gpu::CudaDeviceWorker::synchronize() const {
    hipSetDevice(device_id_);
    hipDeviceSynchronize();
}

template <typename KernelFunc, typename... Args>
void gpu::CudaDeviceWorker::launch_kernel(KernelFunc kernel, dim3 grid_dim, dim3 block_dim, Args... args, size_t shared_mem_size) {
    kernel<<<grid_dim, block_dim, shared_mem_size, stream_>>>(args...);

    // Check for errors after kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA kernel launch failed: ") + hipGetErrorString(err));
    }
}

// Explicit template instantiations (if needed)
template void gpu::CudaDeviceWorker::launch_kernel<void(*)(int*), int*>(void(*)(int*), dim3, dim3, int*, size_t);